#include "hip/hip_runtime.h"
#define GRB_USE_APSPIE
#define COL_MAJOR
//#define private public

#include <iostream>
#include <algorithm>

#include <cstdio>
#include <cstdlib>
#include <hip/hip_runtime_api.h>

#include "graphblas/mmio.hpp"
#include "graphblas/util.hpp"
#include "graphblas/graphblas.hpp"

#define BOOST_TEST_MAIN
#define BOOST_TEST_MODULE spmm_suite

#include <boost/test/included/unit_test.hpp>
#include <test/test.hpp>

struct TestSPMM {
  TestSPMM() :
    argc( boost::unit_test::framework::master_test_suite().argc ),
    argv( boost::unit_test::framework::master_test_suite().argv ) {}

  int argc;
  char **argv;
};

BOOST_AUTO_TEST_SUITE(spmm_suite)

BOOST_FIXTURE_TEST_CASE( spmm1, TestSPMM )
{
  std::vector<graphblas::Index> row_indices;
  std::vector<graphblas::Index> col_indices;
  std::vector<float> values;
	graphblas::Index nrows, ncols, nvals;

	// Read in sparse matrix
  if (argc < 2) {
    fprintf(stderr, "Usage: %s [matrix-market-filename]\n", argv[0]);
    exit(1);
  } else { 
	  readMtx( argv[1], row_indices, col_indices, values, nrows, ncols, nvals );
  }
  //printArray( "row_indices", row_indices );
  //printArray( "col_indices", col_indices );

  graphblas::Matrix<float> a(nrows, ncols);

  graphblas::Index MEM_SIZE = 1000000000;  // 2x4=8GB GPU memory for dense
  graphblas::Index max_ncols = std::min( MEM_SIZE/nrows, ncols );
  if( max_ncols<ncols ) std::cout << "Restricting col to: " << max_ncols <<
      std::endl;

  graphblas::Matrix<float> b(nrows, max_ncols);
  a.build( row_indices, col_indices, values, nvals );
  a.nrows( nrows );
  a.ncols( ncols );
  a.nvals( nvals );
  a.print();
  std::vector<float> denseVal;

  // Row major order
  #ifdef ROW_MAJOR
  for( int i=0; i<nrows; i++ ) {
    for( int j=0; j<max_ncols; j++ ) {
      if( i==j ) denseVal.push_back(1.0);
      else denseVal.push_back(0.0);
    }
  }
  #endif
  // Column major order
  #ifdef COL_MAJOR
  for( int i=0; i<max_ncols; i++ ) {
    for( int j=0; j<nrows; j++ ) {
      denseVal.push_back(1.0);
      //if( i==j ) denseVal.push_back(1.0);
      //else denseVal.push_back(0.0);
    }
  }
  #endif
  b.build( denseVal );
  graphblas::Matrix<float> c(nrows, max_ncols);
  graphblas::Semiring op;

  hipProfilerStart();
  graphblas::mxm<float, float, float>( c, op, a, b );
  hipProfilerStop();

  std::vector<float> out_denseVal;
  c.print();
  c.extractTuples( out_denseVal );
  for( int i=0; i<nvals; i++ ) {
    graphblas::Index row = row_indices[i];
    graphblas::Index col = col_indices[i];
    float            val = values[i];
    if( col<max_ncols ) {
      // Row major order
      #ifdef ROW_MAJOR
      //std::cout << row << " " << col << " " << val << " " << out_denseVal[row*max_ncols+col] << std::endl;
      BOOST_ASSERT( val==out_denseVal[row*max_ncols+col] );
      #endif
      // Column major order
      #ifdef COL_MAJOR
      //std::cout << row << " " << col << " " << val << " " << out_denseVal[col*nrows+row] << std::endl;
      BOOST_ASSERT( val==out_denseVal[col*nrows+row] );
      #endif
    }
}}

BOOST_AUTO_TEST_SUITE_END()

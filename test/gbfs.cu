#define GRB_USE_APSPIE
#define private public

#include <iostream>
#include <algorithm>
#include <string>

#include <cstdio>
#include <cstdlib>

#include <hip/hip_runtime_api.h>

#include <boost/program_options.hpp>

#include "graphblas/graphblas.hpp"
#include "graphblas/algorithm/bfs.hpp"
#include "test/test.hpp"

int main( int argc, char** argv )
{
  std::vector<graphblas::Index> row_indices;
  std::vector<graphblas::Index> col_indices;
  std::vector<float> values;
  graphblas::Index nrows, ncols, nvals;

  // Parse arguments
  bool DEBUG = true;
  bool transpose;
  int  direction;

  // Read in sparse matrix
  if (argc < 2) {
    fprintf(stderr, "Usage: %s [matrix-market-filename]\n", argv[0]);
    exit(1);
  } else { 
    po::variables_map vm;
    parseArgs( argc, argv, vm );
    int directed = vm["directed"].as<int>();
    transpose    = vm["transpose"].as<bool>();
    direction    = vm["direction"].as<int>();
    readMtx( argv[argc-1], row_indices, col_indices, values, nrows, ncols, 
        nvals, directed, DEBUG );
  }

  // Matrix A
  graphblas::Matrix<float> a(nrows, ncols);
  CHECK( a.build(&row_indices, &col_indices, &values, nvals, GrB_NULL) );
  CHECK( a.nrows(&nrows) );
  CHECK( a.ncols(&ncols) );
  CHECK( a.nvals(&nvals) );
  if( DEBUG ) CHECK( a.print() );

  // Vector v
  graphblas::Vector<float> v(nrows);

  // Descriptor desc
  graphblas::Descriptor desc;
  switch( direction )
  {
    case 0:
      CHECK( desc.set(graphblas::GrB_MXVMODE, graphblas::GrB_PUSHPULL) );
      break;
    case 1:
      CHECK( desc.set(graphblas::GrB_MXVMODE, graphblas::GrB_PUSHONLY) );
      break;
    case 2:
      CHECK( desc.set(graphblas::GrB_MXVMODE, graphblas::GrB_PULLONLY) );
      break;
    default:
      std::cout << "Error: incorrect mxvmode selection!\n";
  }

  // Cpu BFS
  CpuTimer bfs_cpu;
  graphblas::Index* h_bfs_cpu = (graphblas::Index*)malloc(nrows*
      sizeof(graphblas::Index));
  int depth = 2000;
  bfs_cpu.Start();
  graphblas::algorithm::bfsCpu( 0, &a, h_bfs_cpu, depth, transpose );
  bfs_cpu.Stop();

  // Warmup
  CpuTimer warmup;
  warmup.Start();
  graphblas::algorithm::bfs(&v, &a, 0, &desc, transpose);
  warmup.Stop();

  std::vector<float> h_bfs_gpu;
  CHECK( v.extractTuples(&h_bfs_gpu, &nrows) );
  BOOST_ASSERT_LIST( h_bfs_cpu, h_bfs_gpu, nrows );
 
  CpuTimer vxm_gpu;
  hipProfilerStart();
  vxm_gpu.Start();
  int NUM_ITER = 1;//0;
  for( int i=0; i<NUM_ITER; i++ )
  {
    graphblas::algorithm::bfs(&v, &a, 0, &desc, transpose);
  }
  hipProfilerStop();
  vxm_gpu.Stop();

  float flop = 0;
  std::cout << "cpu, " << bfs_cpu.ElapsedMillis() << ", \n";
  if( DEBUG ) std::cout << "warmup, " << warmup.ElapsedMillis() << ", " <<
    flop/warmup.ElapsedMillis()/1000000.0 << "\n";
  float elapsed_vxm = vxm_gpu.ElapsedMillis();
  std::cout << "vxm, " << elapsed_vxm/NUM_ITER << "\n";

  CHECK( v.extractTuples(&h_bfs_gpu, &nrows) );
  BOOST_ASSERT_LIST( h_bfs_cpu, h_bfs_gpu, nrows );

  return 0;
}

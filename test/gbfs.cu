#define GRB_USE_APSPIE
#define private public

#include <iostream>
#include <algorithm>
#include <string>

#include <cstdio>
#include <cstdlib>

#include <boost/program_options.hpp>

#include "graphblas/graphblas.hpp"
#include "algorithms/bfs.hpp"
#include "test/test.hpp"

int main( int argc, char** argv )
{
  std::vector<graphblas::Index> row_indices;
  std::vector<graphblas::Index> col_indices;
  std::vector<float> values;
  graphblas::Index nrows, ncols, nvals;

  // Parse arguments
  bool DEBUG = true;

  // Read in sparse matrix
  if (argc < 2) {
    fprintf(stderr, "Usage: %s [matrix-market-filename]\n", argv[0]);
    exit(1);
  } else { 
    readMtx( argv[argc-1], row_indices, col_indices, values, nrows, ncols, 
        nvals, DEBUG );
  }

  // Matrix A
  graphblas::Matrix<float> a(nrows, ncols);
  CHECK( a.build(&row_indices, &col_indices, &values, nvals, GrB_NULL) );
  CHECK( a.nrows(&nrows) );
  CHECK( a.ncols(&ncols) );
  CHECK( a.nvals(&nvals) );
  if( DEBUG ) CHECK( a.print() );

  // Vector v
  graphblas::Vector<float> v(nrows);

  // Warmup
  CpuTimer warmup;
  warmup.Start();
  graphblas::bfs(&v, &a, 0);
  warmup.Stop();
 
  CpuTimer cpu_vxm;
  //hipProfilerStart();
  cpu_vxm.Start();
  int NUM_ITER = 1;//0;
  /*for( int i=0; i<NUM_ITER; i++ )
  {
    graphblas::bfs(&v, &a, 0);
  }*/
  //hipProfilerStop();
  cpu_vxm.Stop();

  float flop = 0;
  if( DEBUG ) std::cout << "warmup, " << warmup.ElapsedMillis() << ", " <<
    flop/warmup.ElapsedMillis()/1000000.0 << "\n";
  float elapsed_vxm = cpu_vxm.ElapsedMillis();
  std::cout << "vxm, " << elapsed_vxm/NUM_ITER << "\n";

  /*c.extractTuples( out_denseVal );
  for( int i=0; i<nvals; i++ )
  {
    graphblas::Index row = row_indices[i];
    graphblas::Index col = col_indices[i];
    float            val = values[i];
    if( col<max_ncols )
    {
      // Row major order
      if( ROW_MAJOR )
      //std::cout << row << " " << col << " " << val << " " << out_denseVal[row*max_ncols+col] << std::endl;
        BOOST_ASSERT( val==out_denseVal[row*max_ncols+col] );
      else
      // Column major order
      //std::cout << row << " " << col << " " << val << " " << out_denseVal[col*nrows+row] << std::endl;
        BOOST_ASSERT( val==out_denseVal[col*nrows+row] );
    }
  }*/
  return 0;
}

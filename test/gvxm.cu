#define GRB_USE_APSPIE
#define private public

#include <iostream>
#include <algorithm>
#include <string>

#include <cstdio>
#include <cstdlib>

#include <boost/program_options.hpp>

#include "graphblas/graphblas.hpp"
#include "test/test.hpp"

int main( int argc, char** argv )
{
  std::vector<graphblas::Index> row_indices;
  std::vector<graphblas::Index> col_indices;
  std::vector<float> values;
  graphblas::Index nrows, ncols, nvals;

  // Parse arguments
  bool debug;
  bool mtxinfo;
  int  directed;
  po::variables_map vm;

  // Read in sparse matrix
  if (argc < 2) {
    fprintf(stderr, "Usage: %s [matrix-market-filename]\n", argv[0]);
    exit(1);
  } else { 
    parseArgs(argc, argv, vm);
    debug    = vm["debug"   ].as<bool>();
    mtxinfo  = vm["mtxinfo" ].as<bool>();
    directed = vm["directed"].as<int>();
    readMtx( argv[argc-1], row_indices, col_indices, values, nrows, ncols, 
        nvals, directed, mtxinfo );
  }

  // Matrix A
  graphblas::Matrix<float> a(nrows, ncols);
  CHECK( a.build(&row_indices, &col_indices, &values, nvals, GrB_NULL) );
  CHECK( a.nrows(&nrows) );
  CHECK( a.ncols(&ncols) );
  CHECK( a.nvals(&nvals) );
  if( debug ) CHECK( a.print() );

  // Vector x
  graphblas::Vector<float> x(nrows);
  CHECK( x.fill(1.0f) );
  CHECK( x.size(&nrows) );
  if( debug ) CHECK( x.print() );

  // Vector y
  graphblas::Vector<float> y(nrows);

  // Descriptor
  graphblas::Descriptor desc;
  CHECK( desc.loadArgs(vm) );
  //CHECK( desc.set(graphblas::GrB_MXVMODE, graphblas::GrB_PUSHONLY) );

  // Warmup
  CpuTimer warmup;
  warmup.Start();
  graphblas::vxm<float, float, float, float>(&y, GrB_NULL, GrB_NULL, 
      graphblas::PlusMultipliesSemiring<float>(), &x, &a, &desc);
  warmup.Stop();
 
  CpuTimer cpu_vxm;
  //hipProfilerStart();
  cpu_vxm.Start();
  int NUM_ITER = 1;//0;
  for( int i=0; i<NUM_ITER; i++ )
  {
    graphblas::vxm<float, float, float, float>( &y, GrB_NULL, GrB_NULL, 
        graphblas::PlusMultipliesSemiring<float>(), &x, &a, &desc );
  }
  //hipProfilerStop();
  cpu_vxm.Stop();

  float flop = 0;
  if( debug ) std::cout << "warmup, " << warmup.ElapsedMillis() << ", " <<
    flop/warmup.ElapsedMillis()/1000000.0 << "\n";
  float elapsed_vxm = cpu_vxm.ElapsedMillis();
  std::cout << "vxm, " << elapsed_vxm/NUM_ITER << "\n";

  if( debug ) y.print();
  return 0;
}

#define GRB_USE_APSPIE
#define private public

#include <iostream>
#include <algorithm>
#include <string>

#include <cstdio>
#include <cstdlib>

#include <boost/program_options.hpp>

#include "graphblas/graphblas.hpp"
#include "test/test.hpp"

int main( int argc, char** argv )
{
  std::vector<graphblas::Index> row_indices;
  std::vector<graphblas::Index> col_indices;
  std::vector<float> values;
  graphblas::Index nrows, ncols, nvals;

  // Parse arguments
  bool DEBUG = true;

  // Read in sparse matrix
  if (argc < 2) {
    fprintf(stderr, "Usage: %s [matrix-market-filename]\n", argv[0]);
    exit(1);
  } else { 
    readMtx( argv[argc-1], row_indices, col_indices, values, nrows, ncols, 
        nvals, DEBUG );
  }

  // Vector mask
  graphblas::Vector<float> m(nrows);
  std::vector<graphblas::Index> m_ind = {1,   2,   3};
  std::vector<float>            m_val = {1.f, 1.f, 1.f};
  CHECK( m.build(&m_ind, &m_val, 3, GrB_NULL) );
  CHECK( m.size(&nrows) );
  if( DEBUG ) CHECK( m.print() );

  // Vector v
  graphblas::Vector<float> v(nrows);
  CHECK( v.fill(-1.f) );
  CHECK( v.setElement(0.f, 1) );
  CHECK( v.size(&nrows) );

  // Descriptor
  graphblas::Descriptor desc;
  //CHECK( desc.set(graphblas::GrB_MASK, graphblas::GrB_SCMP) );

  // Semiring
  graphblas::BinaryOp<float,float,float> GrB_PLUS_FP32;
  GrB_PLUS_FP32.nnew( graphblas::plus<float>() );
  graphblas::BinaryOp<float,float,float> GrB_TIMES_FP32;
  GrB_TIMES_FP32.nnew( graphblas::multiplies<float>() );
  /*graphblas::BinaryOp<float,float,float> GrB_PLUS_FP32;
  GrB_PLUS_FP32.nnew( std::plus<float>() );
  graphblas::BinaryOp<float,float,float> GrB_TIMES_FP32( 
      std::multiplies<float>() );*/
  float A = GrB_PLUS_FP32(3.f,2.f);
  float B = GrB_TIMES_FP32(3.f,2.f);
  //std::cout << A << std::endl;
  //std::cout << B << std::endl;
  graphblas::Monoid  <float> GrB_FP32Add;
  GrB_FP32Add.nnew( GrB_PLUS_FP32, 0.f );
  graphblas::Semiring<float,float,float> GrB_FP32AddMul;
  GrB_FP32AddMul.nnew( GrB_FP32Add, GrB_TIMES_FP32 );

  /*graphblas::BinaryOp GrB_LOR(  graphblas::logical_or() );
  graphblas::BinaryOp GrB_LAND( graphblas::logical_and() );
  graphblas::Monoid   GrB_Lor( GrB_LOR, false );
  graphblas::Semiring GrB_Boolean( GrB_Lor, GrB_LAND );*/

  // Warmup
  CpuTimer warmup;
  warmup.Start();
  graphblas::assign<float, float>(&v, &m, GrB_NULL, (float)1.f, GrB_ALL, nrows, 
      &desc);
  warmup.Stop();
 
  CpuTimer cpu_vxm;
  //hipProfilerStart();
  cpu_vxm.Start();
  int NUM_ITER = 1;//0;
  for( int i=0; i<NUM_ITER; i++ )
  {
    graphblas::assign<float, float>(&v, &m, GrB_NULL, (float)1.f, GrB_ALL,
        nrows, &desc);
  }
  //hipProfilerStop();
  cpu_vxm.Stop();

  float flop = 0;
  if( DEBUG ) std::cout << "warmup, " << warmup.ElapsedMillis() << ", " <<
    flop/warmup.ElapsedMillis()/1000000.0 << "\n";
  float elapsed_vxm = cpu_vxm.ElapsedMillis();
  std::cout << "vxm, " << elapsed_vxm/NUM_ITER << "\n";

  if( DEBUG ) v.print();
  /*c.extractTuples( out_denseVal );
  for( int i=0; i<nvals; i++ )
  {
    graphblas::Index row = row_indices[i];
    graphblas::Index col = col_indices[i];
    float            val = values[i];
    if( col<max_ncols )
    {
      // Row major order
      if( ROW_MAJOR )
      //std::cout << row << " " << col << " " << val << " " << out_denseVal[row*max_ncols+col] << std::endl;
        BOOST_ASSERT( val==out_denseVal[row*max_ncols+col] );
      else
      // Column major order
      //std::cout << row << " " << col << " " << val << " " << out_denseVal[col*nrows+row] << std::endl;
        BOOST_ASSERT( val==out_denseVal[col*nrows+row] );
    }
  }*/
  return 0;
}
